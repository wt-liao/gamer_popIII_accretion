#include "CUAPI.h"

#ifdef GPU



extern real    *d_dt_Array_T;
extern real   (*d_Flu_Array_T)[NCOMP_FLUID][ CUBE(PS1) ];
extern double (*d_Corner_Array_T)[3];
#ifdef GRAVITY
extern real   (*d_Pot_Array_T)[ CUBE(GRA_NXT) ];
#endif




//-------------------------------------------------------------------------------------------------------
// Function    :  CUAPI_MemFree_dt
// Description :  Free the GPU and CPU memory previously allocated by CUAPI_MemAllocate_dt()
//
// Parameter   :  None
//-------------------------------------------------------------------------------------------------------
void CUAPI_MemFree_dt()
{

// free the device memory
   if ( d_dt_Array_T     != NULL )  CUDA_CHECK_ERROR(  hipFree( d_dt_Array_T     )  );
   if ( d_Flu_Array_T    != NULL )  CUDA_CHECK_ERROR(  hipFree( d_Flu_Array_T    )  );
   if ( d_Corner_Array_T != NULL )  CUDA_CHECK_ERROR(  hipFree( d_Corner_Array_T )  );
#  ifdef GRAVITY
   if ( d_Pot_Array_T    != NULL )  CUDA_CHECK_ERROR(  hipFree( d_Pot_Array_T    )  );
#  endif

   d_dt_Array_T     = NULL;
   d_Flu_Array_T    = NULL;
   d_Corner_Array_T = NULL;
#  ifdef GRAVITY
   d_Pot_Array_T    = NULL;
#  endif


// free the host memory allocated by CUDA
   for (int t=0; t<2; t++)
   {
      if ( h_dt_Array_T    [t] != NULL )  CUDA_CHECK_ERROR(  hipHostFree( h_dt_Array_T    [t] )  );
      if ( h_Flu_Array_T   [t] != NULL )  CUDA_CHECK_ERROR(  hipHostFree( h_Flu_Array_T   [t] )  );
      if ( h_Corner_Array_T[t] != NULL )  CUDA_CHECK_ERROR(  hipHostFree( h_Corner_Array_T[t] )  );
#     ifdef GRAVITY
      if ( h_Pot_Array_T   [t] != NULL )  CUDA_CHECK_ERROR(  hipHostFree( h_Pot_Array_T   [t] )  );
#     endif

      h_dt_Array_T    [t] = NULL;
      h_Flu_Array_T   [t] = NULL;
      h_Corner_Array_T[t] = NULL;
#     ifdef GRAVITY
      h_Pot_Array_T   [t] = NULL;
#     endif
   } // for (int t=0; t<2; t++)

} // FUNCTION : CUAPI_MemFree_dt


#endif // #ifdef GPU
