#include "CUAPI.h"
#include "CUFLU.h"

#ifdef GPU



// *******************************************
// ** CUDA stream objects are declared here **
hipStream_t *Stream;
// *******************************************


extern real (*d_Flu_Array_F_In )[FLU_NIN ][ FLU_NXT*FLU_NXT*FLU_NXT ];
extern real (*d_Flu_Array_F_Out)[FLU_NOUT][ PS2*PS2*PS2 ];
extern real (*d_Flux_Array)[9][NFLUX_TOTAL][ PS2*PS2 ];
#ifdef UNSPLIT_GRAVITY
extern real (*d_Pot_Array_USG_F)[ USG_NXT_F*USG_NXT_F*USG_NXT_F ];
#endif
extern double (*d_Corner_Array_F)[3];
#ifdef DUAL_ENERGY
extern char (*d_DE_Array_F_Out)[ PS2*PS2*PS2 ];
#endif

// global memory arrays in different models
#if ( MODEL == HYDRO )
#if ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP  ||  FLU_SCHEME == CTU )
extern real (*d_PriVar)     [NCOMP_TOTAL][ FLU_NXT*FLU_NXT*FLU_NXT ];
extern real (*d_Slope_PPM_x)[NCOMP_TOTAL][ N_SLOPE_PPM*N_SLOPE_PPM*N_SLOPE_PPM ];
extern real (*d_Slope_PPM_y)[NCOMP_TOTAL][ N_SLOPE_PPM*N_SLOPE_PPM*N_SLOPE_PPM ];
extern real (*d_Slope_PPM_z)[NCOMP_TOTAL][ N_SLOPE_PPM*N_SLOPE_PPM*N_SLOPE_PPM ];
extern real (*d_FC_Var_xL)  [NCOMP_TOTAL][ N_FC_VAR*N_FC_VAR*N_FC_VAR ];
extern real (*d_FC_Var_xR)  [NCOMP_TOTAL][ N_FC_VAR*N_FC_VAR*N_FC_VAR ];
extern real (*d_FC_Var_yL)  [NCOMP_TOTAL][ N_FC_VAR*N_FC_VAR*N_FC_VAR ];
extern real (*d_FC_Var_yR)  [NCOMP_TOTAL][ N_FC_VAR*N_FC_VAR*N_FC_VAR ];
extern real (*d_FC_Var_zL)  [NCOMP_TOTAL][ N_FC_VAR*N_FC_VAR*N_FC_VAR ];
extern real (*d_FC_Var_zR)  [NCOMP_TOTAL][ N_FC_VAR*N_FC_VAR*N_FC_VAR ];
extern real (*d_FC_Flux_x)  [NCOMP_TOTAL][ N_FC_FLUX*N_FC_FLUX*N_FC_FLUX ];
extern real (*d_FC_Flux_y)  [NCOMP_TOTAL][ N_FC_FLUX*N_FC_FLUX*N_FC_FLUX ];
extern real (*d_FC_Flux_z)  [NCOMP_TOTAL][ N_FC_FLUX*N_FC_FLUX*N_FC_FLUX ];
#endif // #if ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP  ||  FLU_SCHEME == CTU )

#elif ( MODEL == MHD )
#warning : WAIT MHD !!!

#elif ( MODEL != ELBDM )
#warning : DO YOU WANT TO ADD SOMETHING HERE FOR THE NEW MODEL ??
#endif // MODEL




//-------------------------------------------------------------------------------------------------------
// Function    :  CUAPI_MemAllocate_Fluid
// Description :  Allocate GPU and CPU memory for the fluid solver
//
// Parameter   :  Flu_NPG     : Number of patch groups evaluated simultaneously by GPU for the fluid solver
//                GPU_NStream : Number of CUDA stream objects
//-------------------------------------------------------------------------------------------------------
void CUAPI_MemAllocate_Fluid( const int Flu_NPG, const int GPU_NStream )
{

// determine whether or not to allocate the corner array
   bool AllocateCorner = false;

#  ifdef UNSPLIT_GRAVITY
   if ( OPT__GRAVITY_TYPE == GRAVITY_EXTERNAL  ||  OPT__GRAVITY_TYPE == GRAVITY_BOTH )
      AllocateCorner = true;
#  endif

#  if ( COORDINATE != CARTESIAN )
      AllocateCorner = true;
#  endif


// size of the global memory arrays in all models
   const long Flu_MemSize_F_In  = sizeof(real  )*Flu_NPG*FLU_NIN *FLU_NXT*FLU_NXT*FLU_NXT;
   const long Flu_MemSize_F_Out = sizeof(real  )*Flu_NPG*FLU_NOUT*PS2*PS2*PS2;
   const long Flux_MemSize      = sizeof(real  )*Flu_NPG*9*NFLUX_TOTAL*PS2*PS2;
#  ifdef UNSPLIT_GRAVITY
   const long Pot_MemSize_USG_F = sizeof(real  )*Flu_NPG*USG_NXT_F*USG_NXT_F*USG_NXT_F;
#  endif
   const long Corner_MemSize_F  = ( AllocateCorner ) ? sizeof(double)*Flu_NPG*3 : 0;
#  ifdef DUAL_ENERGY
   const long DE_MemSize_F_Out  = sizeof(char  )*Flu_NPG*PS2*PS2*PS2;
#  endif

// the size of the global memory arrays in different models
#  if   ( MODEL == HYDRO )
#  if ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP  ||  FLU_SCHEME == CTU )
   const long PriVar_MemSize    = Flu_MemSize_F_In;
   const long FC_Var_MemSize    = sizeof(real)*Flu_NPG*NCOMP_TOTAL*N_FC_VAR*N_FC_VAR*N_FC_VAR;
   const long FC_Flux_MemSize   = sizeof(real)*Flu_NPG*NCOMP_TOTAL*N_FC_FLUX*N_FC_FLUX*N_FC_FLUX;

#  if ( LR_SCHEME == PPM )
   const long Slope_PPM_MemSize = sizeof(real)*Flu_NPG*NCOMP_TOTAL*N_SLOPE_PPM*N_SLOPE_PPM*N_SLOPE_PPM;
#  endif

#  endif // #if ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP  ||  FLU_SCHEME == CTU )

#  elif ( MODEL == MHD )
#  warning : WAIT MHD !!!

#  elif ( MODEL != ELBDM )
#  warning : DO YOU WANT TO ADD SOMETHING HERE FOR THE NEW MODEL ??
#  endif // MODEL


// output the total memory requirement
   long TotalSize = Flu_MemSize_F_In + Flu_MemSize_F_Out;

   if ( amr->WithFlux )
   TotalSize += Flux_MemSize;

#  ifdef UNSPLIT_GRAVITY
   TotalSize += Pot_MemSize_USG_F;
#  endif

   if ( AllocateCorner )
   TotalSize += Corner_MemSize_F;

#  ifdef DUAL_ENERGY
   TotalSize += DE_MemSize_F_Out;
#  endif

#  if   ( MODEL == HYDRO )

#  if ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP  ||  FLU_SCHEME == CTU )
   TotalSize += PriVar_MemSize + 6*FC_Var_MemSize + 3*FC_Flux_MemSize;

#  if ( LR_SCHEME == PPM )
   TotalSize += 3*Slope_PPM_MemSize;
#  endif // PPM
#  endif // MHM/MHM_RP/CTU

#  elif ( MODEL == MHD )
#  warning : WAIT MHD !!!

#  elif ( MODEL != ELBDM )
#  warning : DO YOU WANT TO ADD SOMETHING HERE FOR THE NEW MODEL ??
#  endif // MODEL

   if ( MPI_Rank == 0 )
      Aux_Message( stdout, "NOTE : total memory requirement in GPU fluid solver = %ld MB\n", TotalSize/(1<<20) );


// allocate the device memory (in all models)
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Flu_Array_F_In,        Flu_MemSize_F_In        )  );
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Flu_Array_F_Out,       Flu_MemSize_F_Out       )  );

   if ( amr->WithFlux )
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Flux_Array,            Flux_MemSize            )  );

#  ifdef UNSPLIT_GRAVITY
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Pot_Array_USG_F,       Pot_MemSize_USG_F       )  );
#  endif

   if ( AllocateCorner )
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Corner_Array_F,        Corner_MemSize_F        )  );

#  ifdef DUAL_ENERGY
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_DE_Array_F_Out,        DE_MemSize_F_Out        )  );
#  endif


// allocate the device memory (in different models)
#  if   ( MODEL == HYDRO )
#  if ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP  ||  FLU_SCHEME == CTU )
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_PriVar, PriVar_MemSize )  );

#  if ( LR_SCHEME == PPM )
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Slope_PPM_x, Slope_PPM_MemSize )  );
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Slope_PPM_y, Slope_PPM_MemSize )  );
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Slope_PPM_z, Slope_PPM_MemSize )  );
#  endif

   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_FC_Var_xL, FC_Var_MemSize )  );
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_FC_Var_xR, FC_Var_MemSize )  );
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_FC_Var_yL, FC_Var_MemSize )  );
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_FC_Var_yR, FC_Var_MemSize )  );
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_FC_Var_zL, FC_Var_MemSize )  );
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_FC_Var_zR, FC_Var_MemSize )  );

   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_FC_Flux_x, FC_Flux_MemSize )  );
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_FC_Flux_y, FC_Flux_MemSize )  );
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_FC_Flux_z, FC_Flux_MemSize )  );
#  endif // #if ( FLU_SCHEME == MHM  ||  FLU_SCHEME == MHM_RP  ||  FLU_SCHEME == CTU )

#  elif ( MODEL == MHD )
#  warning : WAIT MHD !!!

#  elif ( MODEL != ELBDM )
#  warning : DO YOU WANT TO ADD SOMETHING HERE FOR THE NEW MODEL ??
#  endif // MODEL


// allocate the host memory by CUDA
   for (int t=0; t<2; t++)
   {
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_Flu_Array_F_In [t], Flu_MemSize_F_In        )  );
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_Flu_Array_F_Out[t], Flu_MemSize_F_Out       )  );

      if ( amr->WithFlux )
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_Flux_Array     [t], Flux_MemSize            )  );

#     ifdef UNSPLIT_GRAVITY
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_Pot_Array_USG_F[t], Pot_MemSize_USG_F       )  );
#     endif

      if ( AllocateCorner )
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_Corner_Array_F [t], Corner_MemSize_F        )  );

#     ifdef DUAL_ENERGY
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_DE_Array_F_Out [t], DE_MemSize_F_Out        )  );
#     endif
   } // for (int t=0; t<2; t++)


// create streams
   Stream = new hipStream_t [GPU_NStream];
   for (int s=0; s<GPU_NStream; s++)      CUDA_CHECK_ERROR(  hipStreamCreate( &Stream[s] )  );

} // FUNCTION : CUAPI_MemAllocate_Fluid



#endif // #ifdef GPU
