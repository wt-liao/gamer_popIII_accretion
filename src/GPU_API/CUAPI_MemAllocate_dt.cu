#include "CUAPI.h"

#ifdef GPU



extern real    *d_dt_Array_T;
extern real   (*d_Flu_Array_T)[NCOMP_FLUID][ CUBE(PS1) ];
extern double (*d_Corner_Array_T)[3];
#ifdef GRAVITY
extern real   (*d_Pot_Array_T)[ CUBE(GRA_NXT) ];
#endif




//-------------------------------------------------------------------------------------------------------
// Function    :  CUAPI_MemAllocate_dt
// Description :  Allocate GPU and CPU memory for the dt solver
//
// Parameter   :  dt_NPG : Number of patch groups evaluated simultaneously by GPU for the dt solver
//-------------------------------------------------------------------------------------------------------
void CUAPI_MemAllocate_dt( const int dt_NPG )
{

// size of the global memory arrays
   const int  dt_NP            = 8*dt_NPG;
   const long dt_MemSize_T     = sizeof(real  )*dt_NP;
   const long Flu_MemSize_T    = sizeof(real  )*dt_NP*NCOMP_FLUID*CUBE(PS1);
   const long Corner_MemSize_T = sizeof(double)*dt_NP*3;
#  ifdef GRAVITY
   const long Pot_MemSize_T    = sizeof(real  )*dt_NP*CUBE(GRA_NXT);
#  endif


// output the total memory requirement
   long TotalSize = dt_MemSize_T + Flu_MemSize_T + Corner_MemSize_T;
#  ifdef GRAVITY
   TotalSize += Pot_MemSize_T;
#  endif

   if ( MPI_Rank == 0 )
      Aux_Message( stdout, "NOTE : total memory requirement in GPU dt solver = %ld MB\n", TotalSize/(1<<20) );


// allocate the device memory
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_dt_Array_T,               dt_MemSize_T            )  );
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Flu_Array_T,              Flu_MemSize_T           )  );
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Corner_Array_T,           Corner_MemSize_T        )  );
#  ifdef GRAVITY
   CUDA_CHECK_ERROR(  hipMalloc( (void**) &d_Pot_Array_T,              Pot_MemSize_T           )  );
#  endif


// allocate the host memory by CUDA
   for (int t=0; t<2; t++)
   {
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_dt_Array_T    [t], dt_MemSize_T            )  );
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_Flu_Array_T   [t], Flu_MemSize_T           )  );
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_Corner_Array_T[t], Corner_MemSize_T        )  );
#     ifdef GRAVITY
      CUDA_CHECK_ERROR(  hipHostMalloc( (void**) &h_Pot_Array_T   [t], Pot_MemSize_T           )  );
#     endif
   } // for (int t=0; t<2; t++)

} // FUNCTION : CUAPI_MemAllocate_dt



#endif // #ifdef GPU
