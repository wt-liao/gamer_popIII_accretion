#ifdef __HIPCC__
#include "Macro.h"
#else
#include "GAMER.h"
#endif
#include "CUPOT.h"

#ifdef GRAVITY




//-----------------------------------------------------------------------------------------
// Function    :  CUPOT_ExternalPot / CPU_ExternalPot
// Description :  Calculate the external potential at the given coordinates and time
//
// Note        :  1. This function will be invoked by both CPU and GPU
//                2. The auxiliary array "UserArray" is set by "Init_ExternalPot_Ptr", which
//                   points to "Init_ExternalPot()" by default but may be overwritten by various
//                   test problem initializers
//                3. By default we assume
//                     UserArray[0] = x coordinate of the external acceleration center
//                     UserArray[1] = y ...
//                     UserArray[2] = z ..
//                     UserArray[3] = gravitational_constant*point_source_mass
//                   --> but one can easily modify this file to change the default behavior
//                4. Currently it does not support the soften length
//
// Parameter   :  X/Y/Z     : Target spatial coordinates in the adopted coordinate system
//                Time      : Current physical time
//                UserArray : User-provided auxiliary array (set by "Init_ExternalPot_Ptr")
//
// Return      :  External potential
//-----------------------------------------------------------------------------------------
#ifdef __HIPCC__
__device__
real CUPOT_ExternalPot( const double X, const double Y, const double Z, const double Time, const double UserArray[] )
#else
real   CPU_ExternalPot( const double X, const double Y, const double Z, const double Time, const double UserArray[] )
#endif
{

   const real   GM     = (real)UserArray[3];
#  if ( COORDINATE == CARTESIAN )
   const double Cen[3] = { UserArray[0], UserArray[1], UserArray[2] };
   const real   dx     = (real)(X - Cen[0]);
   const real   dy     = (real)(Y - Cen[1]);
   const real   dz     = (real)(Z - Cen[2]);
   const real   _r     = (real)1.0/SQRT( dx*dx + dy*dy + dz*dz );
#  else
   const real  _r      = (real)1.0/X;
#  endif

   return -GM*_r;

} // FUNCTION : CUPOT_ExternalPot // CPU_ExternalPot



#endif // #ifdef GRAVITY
