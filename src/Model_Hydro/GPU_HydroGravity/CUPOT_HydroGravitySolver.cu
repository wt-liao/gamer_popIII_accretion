#include "hip/hip_runtime.h"
#include "Macro.h"
#include "CUPOT.h"

#if ( defined GPU  &&  MODEL == HYDRO  &&  defined GRAVITY )



#include "../../SelfGravity/GPU_Gravity/CUPOT_ExternalAcc.cu"
#define GRA_NTHREAD  ( PATCH_SIZE*PATCH_SIZE*GRA_BLOCK_SIZE_Z )

// variables reside in constant memory
__constant__ double ExtAcc_AuxArray_d[EXT_ACC_NAUX_MAX];




//-------------------------------------------------------------------------------------------------------
// Function    :  CUPOT_HydroGravitySolver_SetConstMem
// Description :  Set the constant memory used by CUPOT_HydroGravitySolver
//
// Note        :  Adopt the suggested approach for CUDA version >= 5.0
//
// Parameter   :  None
//
// Return      :  0/-1 : successful/failed
//---------------------------------------------------------------------------------------------------
int CUPOT_HydroGravitySolver_SetConstMem( double ExtAcc_AuxArray_h[] )
{

   if (  hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL( ExtAcc_AuxArray_d), ExtAcc_AuxArray_h, EXT_ACC_NAUX_MAX*sizeof(double),
                                            0, hipMemcpyHostToDevice)  )
      return -1;

   else
      return 0;

} // FUNCTION : CUPOT_HydroGravitySolver_SetConstMem



//-------------------------------------------------------------------------------------------------------
// Function    :  CUPOT_HydroGravitySolver
// Description :  GPU gravity solver which advances the momentum and energy density of a group of patches
//                by gravitational acceleration (including the external gravity)
//
// Note        :  1. Prefix "g" for pointers pointing to the "Global" memory space
//                   Prefix "s" for pointers pointing to the "Shared" memory space
//                2. Currently this function does NOT ensure the consistency between Etot-Ekin and
//                   the dual-energy variable (either internal energy of entropy)
//                   --> This consistency breaks only for cells with the dual-energy status labelled
//                       as DE_UPDATED_BY_ETOT_GRA
//                   --> We restore this consistency in Gra_Close()
//
// Parameter   :  g_Flu_Array_New : Global memory array to store the input and output fluid variables
//                g_Pot_Array_New : Global memory array storing the input potential for evaluating the
//                                  gravitational acceleration
//                g_Corner_Array  : Global memory array storing the physical corner coordinates of each patch
//                g_Pot_Array_USG : Global memory array storing the prepared potential          for UNSPLIT_GRAVITY (at the previous step)
//                g_Flu_Array_USG : Global memory array storing the prepared density + momentum for UNSPLIT_GRAVITY (at the previous step)
//                g_DE_Array      : Global memory array storing the dual-energy status (for both input and output)
//                Gra_Const       : 3-P stencil : -dt / ( 2*dh)
//                                  5-P stencil : -dt / (12*dh)
//                P5_Gradient     : Use 5-points stecil to evaluate the potential gradient
//                GravityType     : Types of gravity --> self-gravity, external gravity, both
//                TimeNew         : Physical time at the current  step (for the external gravity solver)
//                TimeOld         : Physical time at the previous step (for the external gravity solver in UNSPLIT_GRAVITY)
//                dt              : Evolution time-step (for the external gravity solver)
//                dh              : Grid size (for the external gravity solver)
//                MinEint         : Minimum allowed internal energy (== MIN_PRES / (GAMMA-1))
//---------------------------------------------------------------------------------------------------
__global__ void CUPOT_HydroGravitySolver(       real g_Flu_Array_New[][GRA_NIN][ PS1*PS1*PS1 ],
                                          const real g_Pot_Array_New[][ GRA_NXT*GRA_NXT*GRA_NXT ],
                                          const double g_Corner_Array[][3],
                                          const real g_Pot_Array_USG[][ USG_NXT_G*USG_NXT_G*USG_NXT_G ],
                                          const real g_Flu_Array_USG[][GRA_NIN-1][ PS1*PS1*PS1 ],
                                                char g_DE_Array[][ PS1*PS1*PS1 ],
                                          const real Gra_Const, const bool P5_Gradient, const OptGravityType_t GravityType,
                                          const double TimeNew, const double TimeOld, const real dt, const real dh, const real MinEint )
{

   const uint bx     = blockIdx.x;
   const uint tx     = threadIdx.x;
   const uint ty     = threadIdx.y;
   const uint tz     = threadIdx.z;
   const uint ID     = __umul24( tz, PS1*PS1 ) + __umul24( ty, PS1 ) + tx;
   const uint NSlice = GRA_BLOCK_SIZE_Z;

   uint g_idx        = ID;
   uint s_idx_new    =   __umul24( GRA_GHOST_SIZE+tz, GRA_NXT*GRA_NXT )
                       + __umul24( GRA_GHOST_SIZE+ty, GRA_NXT   ) + (GRA_GHOST_SIZE+tx);

   uint   ip1_new, jp1_new, kp1_new, im1_new, jm1_new, km1_new, t;
   uint   ip2_new, jp2_new, kp2_new, im2_new, jm2_new, km2_new;
   real   Acc_new[3], px_new, py_new, pz_new, rho_new;
   real   Eint_in, Ek_out, Etot_in, Etot_out, _rho2;
   double x, y, z;

   __shared__ real s_Pot_new[GRA_NXT*GRA_NXT*GRA_NXT];

#  ifdef UNSPLIT_GRAVITY
   uint s_idx_old    =   __umul24( USG_GHOST_SIZE+tz, USG_NXT_G*USG_NXT_G )
                       + __umul24( USG_GHOST_SIZE+ty, USG_NXT_G ) + (USG_GHOST_SIZE+tx);

   uint ip1_old, jp1_old, kp1_old, im1_old, jm1_old, km1_old;
   uint ip2_old, jp2_old, kp2_old, im2_old, jm2_old, km2_old;
   real Acc_old[3], px_old, py_old, pz_old, rho_old;

   __shared__ real s_Pot_old[USG_NXT_G*USG_NXT_G*USG_NXT_G];
#  endif


// set the physical coordinates of each cell for the external gravity solver
   if ( GravityType == GRAVITY_EXTERNAL  ||  GravityType == GRAVITY_BOTH )
   {
      x = g_Corner_Array[bx][0] + (double)(tx*dh);
      y = g_Corner_Array[bx][1] + (double)(ty*dh);
      z = g_Corner_Array[bx][2] + (double)(tz*dh);
   }


// load the potential from the global memory to the shared memory
   if ( GravityType == GRAVITY_SELF  ||  GravityType == GRAVITY_BOTH )
   {
      t = ID;
      do {  s_Pot_new[t] = g_Pot_Array_New[bx][t];   t += GRA_NTHREAD; }  while ( t < CUBE(GRA_NXT) );

#     ifdef UNSPLIT_GRAVITY
      t = ID;
      do {  s_Pot_old[t] = g_Pot_Array_USG[bx][t];   t += GRA_NTHREAD; }  while ( t < CUBE(USG_NXT_G) );
#     endif
   }

   __syncthreads();


   for (uint Slice=tz; Slice<PS1; Slice+=NSlice)
   {
      ip1_new = s_idx_new + 1;
      jp1_new = s_idx_new + GRA_NXT;
      kp1_new = s_idx_new + GRA_NXT*GRA_NXT;
      im1_new = s_idx_new - 1;
      jm1_new = s_idx_new - GRA_NXT;
      km1_new = s_idx_new - GRA_NXT*GRA_NXT;

#     ifdef UNSPLIT_GRAVITY
      ip1_old = s_idx_old + 1;
      jp1_old = s_idx_old + USG_NXT_G;
      kp1_old = s_idx_old + USG_NXT_G*USG_NXT_G;
      im1_old = s_idx_old - 1;
      jm1_old = s_idx_old - USG_NXT_G;
      km1_old = s_idx_old - USG_NXT_G*USG_NXT_G;
#     endif

      if ( P5_Gradient )
      {
         ip2_new = s_idx_new + 2;
         jp2_new = s_idx_new + 2*GRA_NXT;
         kp2_new = s_idx_new + 2*GRA_NXT*GRA_NXT;
         im2_new = s_idx_new - 2;
         jm2_new = s_idx_new - 2*GRA_NXT;
         km2_new = s_idx_new - 2*GRA_NXT*GRA_NXT;

#        ifdef UNSPLIT_GRAVITY
         ip2_old = s_idx_old + 2;
         jp2_old = s_idx_old + 2*USG_NXT_G;
         kp2_old = s_idx_old + 2*USG_NXT_G*USG_NXT_G;
         im2_old = s_idx_old - 2;
         jm2_old = s_idx_old - 2*USG_NXT_G;
         km2_old = s_idx_old - 2*USG_NXT_G*USG_NXT_G;
#        endif
      } // if ( P5_Gradient )


//    1. evaluate the gravitational acceleration
      Acc_new[0] = (real)0.0;
      Acc_new[1] = (real)0.0;
      Acc_new[2] = (real)0.0;

#     ifdef UNSPLIT_GRAVITY
      Acc_old[0] = (real)0.0;
      Acc_old[1] = (real)0.0;
      Acc_old[2] = (real)0.0;
#     endif

//    1.1 external gravity
      if ( GravityType == GRAVITY_EXTERNAL  ||  GravityType == GRAVITY_BOTH )
      {
         CUPOT_ExternalAcc( Acc_new, x, y, z, TimeNew, ExtAcc_AuxArray_d );
         for (int d=0; d<3; d++)    Acc_new[d] *= dt;

#        ifdef UNSPLIT_GRAVITY
         CUPOT_ExternalAcc( Acc_old, x, y, z, TimeOld, ExtAcc_AuxArray_d );
         for (int d=0; d<3; d++)    Acc_old[d] *= dt;
#        endif
      }

//    1.2 self-gravity
      if ( GravityType == GRAVITY_SELF  ||  GravityType == GRAVITY_BOTH )
      {
         if ( P5_Gradient )   // 5-point gradient
         {
            Acc_new[0] += Gra_Const*( - s_Pot_new[ip2_new] + (real)8.0*s_Pot_new[ip1_new] - (real)8.0*s_Pot_new[im1_new] + s_Pot_new[im2_new] );
            Acc_new[1] += Gra_Const*( - s_Pot_new[jp2_new] + (real)8.0*s_Pot_new[jp1_new] - (real)8.0*s_Pot_new[jm1_new] + s_Pot_new[jm2_new] );
            Acc_new[2] += Gra_Const*( - s_Pot_new[kp2_new] + (real)8.0*s_Pot_new[kp1_new] - (real)8.0*s_Pot_new[km1_new] + s_Pot_new[km2_new] );

#           ifdef UNSPLIT_GRAVITY
            Acc_old[0] += Gra_Const*( - s_Pot_old[ip2_old] + (real)8.0*s_Pot_old[ip1_old] - (real)8.0*s_Pot_old[im1_old] + s_Pot_old[im2_old] );
            Acc_old[1] += Gra_Const*( - s_Pot_old[jp2_old] + (real)8.0*s_Pot_old[jp1_old] - (real)8.0*s_Pot_old[jm1_old] + s_Pot_old[jm2_old] );
            Acc_old[2] += Gra_Const*( - s_Pot_old[kp2_old] + (real)8.0*s_Pot_old[kp1_old] - (real)8.0*s_Pot_old[km1_old] + s_Pot_old[km2_old] );
#           endif
         }

         else                 // 3-point gradient
         {
            Acc_new[0] += Gra_Const*( s_Pot_new[ip1_new] - s_Pot_new[im1_new] );
            Acc_new[1] += Gra_Const*( s_Pot_new[jp1_new] - s_Pot_new[jm1_new] );
            Acc_new[2] += Gra_Const*( s_Pot_new[kp1_new] - s_Pot_new[km1_new] );

#           ifdef UNSPLIT_GRAVITY
            Acc_old[0] += Gra_Const*( s_Pot_old[ip1_old] - s_Pot_old[im1_old] );
            Acc_old[1] += Gra_Const*( s_Pot_old[jp1_old] - s_Pot_old[jm1_old] );
            Acc_old[2] += Gra_Const*( s_Pot_old[kp1_old] - s_Pot_old[km1_old] );
#           endif
         }
      } // if ( GravityType == GRAVITY_SELF  ||  GravityType == GRAVITY_BOTH )


//    2. advance the fluid
#     ifdef UNSPLIT_GRAVITY

      rho_new = g_Flu_Array_New[bx][DENS][g_idx];
      rho_old = g_Flu_Array_USG[bx][DENS][g_idx];
      px_new  = g_Flu_Array_New[bx][MOMX][g_idx];
      px_old  = g_Flu_Array_USG[bx][MOMX][g_idx];
      py_new  = g_Flu_Array_New[bx][MOMY][g_idx];
      py_old  = g_Flu_Array_USG[bx][MOMY][g_idx];
      pz_new  = g_Flu_Array_New[bx][MOMZ][g_idx];
      pz_old  = g_Flu_Array_USG[bx][MOMZ][g_idx];

//    backup the original internal energy so that we can restore it later if necessary
      _rho2   = (real)0.5/rho_new;
      Etot_in = g_Flu_Array_New[bx][ENGY][g_idx];
      Eint_in = Etot_in - _rho2*( SQR(px_new) + SQR(py_new) + SQR(pz_new) );

//    update the momentum density
      px_new += (real)0.5*( rho_old*Acc_old[0] + rho_new*Acc_new[0] );
      py_new += (real)0.5*( rho_old*Acc_old[1] + rho_new*Acc_new[1] );
      pz_new += (real)0.5*( rho_old*Acc_old[2] + rho_new*Acc_new[2] );

      g_Flu_Array_New[bx][MOMX][g_idx] = px_new;
      g_Flu_Array_New[bx][MOMY][g_idx] = py_new;
      g_Flu_Array_New[bx][MOMZ][g_idx] = pz_new;

//    record the updated kinematic energy density
      Ek_out = _rho2*( SQR(px_new) + SQR(py_new) + SQR(pz_new) );

//    update the total energy density
#     ifdef DUAL_ENERGY

//    for the unsplitting method with the dual-energy formalism, we correct the **total energy density**
//    only if the dual-energy status != DE_UPDATED_BY_DUAL
//    --> for (a) DE_UPDATED_BY_DUAL     --> Eint has been updated by the dual-energy variable
//            (b) DE_UPDATED_BY_MIN_PRES --> Eint has been set to the minimum threshold
//    --> currently for (b) we still update the total energy density

      if ( g_DE_Array[bx][g_idx] == DE_UPDATED_BY_DUAL )
      {
//       fix the internal energy and the dual-energy variable
         Etot_out = Eint_in + Ek_out;
      }

      else
      {
//       update the total energy, where internal energy and dual-energy variable may change as well
         Etot_out = Etot_in + (real)0.5*( px_old*Acc_old[0] + py_old*Acc_old[1] + pz_old*Acc_old[2] +
                                          px_new*Acc_new[0] + py_new*Acc_new[1] + pz_new*Acc_new[2] );

//       check the minimum internal energy
//       (a) if the updated internal energy is greater than the threshold, set the dual-energy status == DE_UPDATED_BY_ETOT_GRA
         if ( Etot_out - Ek_out >= MinEint )
            g_DE_Array[bx][g_idx] = DE_UPDATED_BY_ETOT_GRA;

//       (b) otherwise restore the original internal energy and keep the original dual-energy status
         else
            Etot_out = Eint_in + Ek_out;
      }

#     else // # ifdef DUAL_ENERGY

//    for the unsplitting method without the dual-energy formalism, we always correct the total energy density
//    instead of the kinematic energy density
//    --> internal energy may change
//    --> we must check the minimum internal after this update
      Etot_out = Etot_in + (real)0.5*( px_old*Acc_old[0] + py_old*Acc_old[1] + pz_old*Acc_old[2] +
                                       px_new*Acc_new[0] + py_new*Acc_new[1] + pz_new*Acc_new[2] );

//    check the minimum internal energy
//    --> restore to the original internal energy if the updated value becomes smaller than the threshold
      if ( Etot_out - Ek_out < MinEint )
         Etot_out = Eint_in + Ek_out;

#     endif // #ifdef DUAL_ENERGY ... else ...


#     else // #ifdef UNSPLIT_GRAVITY

      rho_new = g_Flu_Array_New[bx][DENS][g_idx];
      px_new  = g_Flu_Array_New[bx][MOMX][g_idx];
      py_new  = g_Flu_Array_New[bx][MOMY][g_idx];
      pz_new  = g_Flu_Array_New[bx][MOMZ][g_idx];

//    backup the original internal energy so that we can restore it later if necessary
      _rho2   = (real)0.5/rho_new;
      Etot_in = g_Flu_Array_New[bx][ENGY][g_idx];
      Eint_in = Etot_in - _rho2*( SQR(px_new) + SQR(py_new) + SQR(pz_new) );

//    update the momentum density
      px_new += rho_new*Acc_new[0];
      py_new += rho_new*Acc_new[1];
      pz_new += rho_new*Acc_new[2];

      g_Flu_Array_New[bx][MOMX][g_idx] = px_new;
      g_Flu_Array_New[bx][MOMY][g_idx] = py_new;
      g_Flu_Array_New[bx][MOMZ][g_idx] = pz_new;

//    for the splitting method, we ensure that the internal energy is unchanged
      Ek_out   = _rho2*( SQR(px_new) + SQR(py_new) + SQR(pz_new) );
      Etot_out = Eint_in + Ek_out;

#     endif // #ifdef UNSPLIT_GRAVITY ... else ...


//    store the updated total energy density back to the global memory
      g_Flu_Array_New[bx][ENGY][g_idx] = Etot_out;


//    update target cell indices
      s_idx_new += NSlice*SQR(GRA_NXT);
#     ifdef UNSPLIT_GRAVITY
      s_idx_old += NSlice*SQR(USG_NXT_G);
#     endif
      g_idx     += NSlice*SQR(PS1);
      z         += NSlice*dh;
   } // for (uint Slice=tz; Slice<PS1; Slice+=NSlice)

} // FUNCTION : CUPOT_HydroGravitySolver



#endif // #if ( defined GPU  &&  MODEL == HYDRO  &&  defined GRAVITY )
